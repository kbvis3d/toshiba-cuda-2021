#include "hip/hip_runtime.h"
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int main(void)
{
    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the input and output vectors
    float* A, * B, * C;
    hipMallocManaged(&A, size);
    hipMallocManaged(&B, size);
    hipMallocManaged(&C, size);

    // Initialize the input vectors
    for (int i = 0; i < numElements; ++i)
    {
        A[i] = rand()/(float)RAND_MAX;
        B[i] = rand()/(float)RAND_MAX;
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, numElements);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(A[i] + B[i] - C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
    hipFree(A);
    hipFree(B);
    hipFree(C);

    printf("Done\n");
    return 0;
}

